#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

namespace {
// Blocks: <n // 1024>, Threads: <min(n, 1024)>
__global__ void adan2nd_fp32_accum(
    int32_t n,
    const half *g,            // (n)
    const half *neg_pre_g,    // (n)
    // const half *hessian_est,  // (n), could be nullptr
    float *exp_avg,           // (n)
    float *exp_avg_diff,      // (n)
    float *exp_avg_sq,        // (n)
    float *param,             // (n)
    half *param_h,            // (n)
    float beta1,
    float beta2,
    // float beta3,
    float eps,
    float lr,
    float rho,
    float scale,
    float weight_decay,
    float bias_correction1,
    float bias_correction2,
    float bias_correction3
) {
    int32_t global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id >= n) return;

    float local_g = __half2float(g[global_id]);  // real_g * scale
    float update = local_g + __half2float(neg_pre_g[global_id]);  // real_g * scale

    // Update exp_avg and exp_avg_diff
    exp_avg[global_id] = beta1 * exp_avg[global_id] + (1 - beta1) * local_g;
    exp_avg_diff[global_id] = beta2 * exp_avg_diff[global_id] + (1 - beta2) * update;

    // // Update exp_avg_sq
    // if (hessian_est != nullptr) {
    //     // real_est * scale
    //     exp_avg_sq[global_id] = beta3 * exp_avg_sq[global_id] + (1 - beta3) * __half2float(hessian_est[global_id]);
    // }

    // Update parameters
    float denom = exp_avg_sq[global_id] / bias_correction3 * rho + eps * scale;
    float step_size_diff = lr * beta2 / bias_correction2;
    float step_size = lr / bias_correction1;

    
    param[global_id] -= max(min(step_size * exp_avg[global_id] / denom
                        + step_size_diff * exp_avg_diff[global_id] / denom, lr), -lr);
    param[global_id] /= (1 + lr * weight_decay);

    param_h[global_id] = __float2half(param[global_id]);
}

}

void adan2nd_launcher(
    const torch::Tensor &param_fp32,
    const torch::Tensor &param_fp16,
    const torch::Tensor &g_fp16,
    const torch::Tensor &neg_pre_g_fp16,
    // const std::optional<torch::Tensor> &hessian_est_fp16,
    const torch::Tensor &exp_avg_fp32,
    const torch::Tensor &exp_avg_diff_fp32,
    const torch::Tensor &exp_avg_sq_fp32,
    float beta1, float beta2, //float beta3, 
    float eps, float lr, float rho,
    float scale, 
    float weight_decay, 
    float bias_correction1, 
    float bias_correction2, 
    float bias_correction3
) {
    int32_t n = param_fp32.numel();
    if (n <= 0) return;

    auto g_ptr = reinterpret_cast<half*>(g_fp16.data_ptr<at::Half>());
    auto neg_pre_g_ptr = reinterpret_cast<half*>(neg_pre_g_fp16.data_ptr<at::Half>());
    auto exp_avg_ptr = exp_avg_fp32.data_ptr<float>();
    auto exp_avg_diff_ptr = exp_avg_diff_fp32.data_ptr<float>();
    auto exp_avg_sq_ptr = exp_avg_sq_fp32.data_ptr<float>();
    auto param_ptr = param_fp32.data_ptr<float>();
    auto param_h_ptr = reinterpret_cast<half*>(param_fp16.data_ptr<at::Half>());

    // half* hessian_est_ptr = nullptr;
    // if (hessian_est_fp16.has_value()) {
    //     hessian_est_ptr = reinterpret_cast<half*>(hessian_est_fp16.value().data_ptr<at::Half>());
    // }

    int32_t threads = 1024;
    dim3 block_size = dim3(threads, 1, 1);
    dim3 grid_size = dim3((n + threads - 1) / threads, 1, 1);

    auto stream = at::cuda::getCurrentCUDAStream();

    adan2nd_fp32_accum<<<grid_size, block_size, 0, stream.stream()>>>(
        n, g_ptr, neg_pre_g_ptr, exp_avg_ptr, exp_avg_diff_ptr, 
        exp_avg_sq_ptr, param_ptr, param_h_ptr,
        beta1, beta2, eps, lr, rho,
        scale, weight_decay,
        bias_correction1, bias_correction2, bias_correction3
    );
}
